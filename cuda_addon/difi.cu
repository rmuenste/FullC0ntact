#include "hip/hip_runtime.h"
#include <stdio.h>
#include <common.h>
#include <perftimer.h>
#include <cfloat>
#include <difi.cuh>
#include <cutil_math.cuh>
#include <aabb3.h>
#include <boundingvolumetree3.h>
#include <thrust/tuple.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>
#include <cmath>


struct printf_functor
{
  __host__ __device__
  void operator()(int x)
  {
      // note that using printf in a __device__ function requires
      // code compiled for a GPU with compute capability 2.0 or
      // higher (nvcc --arch=sm_20)
      printf("%d\n", x);
  }
};

void test_iters()
{
  thrust::device_vector<int> d_vec(3);
  d_vec[0] = 0; d_vec[1] = 1; d_vec[2] = 2;
  thrust::for_each(thrust::device, d_vec.begin(), d_vec.end(), printf_functor());
}

void cudaCheckError(const char *message, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
  {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s User error message: %s\n",
        file, line, hipGetErrorString(err), message);
    exit(-1);
  }

}

inline void cuErr(hipError_t status, const char *file, const int line)
{
  if(status != hipSuccess) {
    std::cerr << "Cuda API error: ";
    std::cerr << hipGetErrorString(status);
    std::cerr << " at line " << line;
    std::cerr << " of file " << file << std::endl;
    std::exit(EXIT_FAILURE); 
  }
}

int g_triangles;
int g_verticesGrid;

const int NN = 100 * 1000;

Model3D *g_model;

triangle *d_triangles;
vector3 *d_vertices;
vector3 *d_vertices_grid;

AABB3f *d_boundingBoxes;

vector3 *d_vertexCoords;
vector3 *d_normals;
vector3 *d_contactPoints;      

float *d_distance_map;

int *d_inout;
real *d_distance;


#include "bvh.cuh"
#include "intersection.cuh"
#include "distance.cuh"


#include "distancemap.cuh"
#include "collidermeshmesh.cuh"
#include "uniformgrid.cuh"
#include "unit_tests.cuh"

BVHNode<float> *d_nodes;

DMap *d_map;

DistanceMap<float,gpu> *d_map_gpu;

UniformGrid<float,ElementCell,VertexTraits<float>,gpu> *d_unigrid_gpu;

//HashGrid<float, i3d::gpu> *d_hashGrid;
//ParticleWorld<float, i3d::gpu> *d_particleWorld;

std::vector< DistanceMap<float, gpu>* >  d_maps_gpu;

#include "auxiliary_functions.cuh"

__global__ void outputHashGrid(HashGrid<float,gpu> *g)
{
  g->outputInfo();
}

void hashgrid_sort()
{
  //return;
}

void copy_data(i3d::HashGrid<float, i3d::cpu> &hg, 
               i3d::ParticleWorld<float, i3d::cpu> &pw,
               std::vector<float> &pos)
{

  cudaCheck(hipMemcpy(pos.data(),
                       pw.pos_,
                       pw.size_ * 4 * sizeof(float),
                       hipMemcpyDeviceToHost)); 

}

__global__ void hashgrid_size(HashGrid<float,gpu> *g)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  printf("HashGrid size = %i\n",g->size_);

  printf("HashGrid number of cells = %i\n",g->numCells_);

  printf("Grid X Y Z = %i %i %i\n",g->gridx_, g->gridy_, g->gridz_);

  printf("Cell size = [%f %f %f]\n" ,g->cellSize_.x
                                    ,g->cellSize_.y
                                    ,g->cellSize_.z);

  printf("Origin = [%f %f %f]\n" ,g->origin_.x
                                 ,g->origin_.y
                                 ,g->origin_.z);

  for(int i(0); i < g->size_; ++i)
  {
    g->hashEntries_[i]=20-i;
    g->particleIndices_[i]=20-i;
    printf("particleIndices_[%i]=%i\n",i, g->particleIndices_[i]);
  }

}

__global__ void output_sorted(HashGrid<float, gpu> *g, ParticleWorld<float,gpu> *w)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  float4 *pos_ = (float4*)w->pos_;  
  float4 *pos_sorted = (float4*)w->sortedPos_;  


  for (int i(0); i < g->size_; ++i)
  {
    printf(" hashEntries_[%i]=%i particleIndices_[%i]=%i \n", i, g->hashEntries_[i], i, g->particleIndices_[i]);

    printf("pos_unsorted = [%f %f %f]\n" ,pos_[i].x
                                         ,pos_[i].y
                                         ,pos_[i].z);

    printf("pos_sorted = [%f %f %f]\n" ,pos_sorted[i].x
                                       ,pos_sorted[i].y
                                       ,pos_sorted[i].z);

  }

}

__global__ void test_particleworld(ParticleWorld<float,gpu> *w)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  printf("Spring = %f\n",w->params_->spring_);
  printf("Damping = %f\n",w->params_->damping_);
  printf("Shear = %f\n",w->params_->shear_);
  printf("Attraction = %f\n",w->params_->attraction_);
  printf("Global dampening = %f\n",w->params_->globalDamping_);
  printf("Particle radius = %f\n",w->params_->particleRadius_);
  printf("Gravity = [%f %f %f]\n",w->params_->gravity_.x
                                 ,w->params_->gravity_.y
                                 ,w->params_->gravity_.z);

  for(int i(0); i < w->size_*4; i+=4)
  {
    printf("pos[%i] = [%f %f %f]\n",i/4
                                   ,w->pos_[i]
                                   ,w->pos_[i+1]
                                   ,w->pos_[i+2]
                                   );

    printf("vel[%i] = [%f %f %f]\n",i/4
                                   ,w->vel_[i]
                                   ,w->vel_[i+1]
                                   ,w->vel_[i+2]
                                   );
  }

}

__global__ void output_cellstart(HashGrid<float, gpu> *g)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  for (int i(0); i < g->size_; ++i)
  {
    //printf(" cellStart_[%i]=%i cellEnd_[%i]=%i \n", i, g->cellStart_[i], i, g->cellEnd_[i]);
    printf("hash=%i cellStart_[%i]=%i cellEnd_[%i]=%i \n", g->hashEntries_[i], g->hashEntries_[i], 
                                                    g->cellStart_[g->hashEntries_[i]],
                                                    g->hashEntries_[i], 
                                                    g->cellEnd_[g->hashEntries_[i]]);
    
  }

}


//void test_hashgrid(HashGrid<float, cpu> &hg, ParticleWorld<float, cpu> &pw,
//    WorldParameters &params)
//{
//  hg.size_ = 10;
//  pw.size_ = hg.size_;
//
//  hg.cellSize_.x = 2.0f * pw.params_->particleRadius_; 
//  hg.cellSize_.y = 2.0f * pw.params_->particleRadius_; 
//  hg.cellSize_.z = 2.0f * pw.params_->particleRadius_; 
//
//  hg.origin_   = pw.params_->origin_;
//
//  hg.gridx_ = pw.params_->gridx_;
//  hg.gridy_ = pw.params_->gridy_;
//  hg.gridz_ = pw.params_->gridz_;
//
//  hg.numCells_ = hg.gridx_ * hg.gridy_ * hg.gridz_;  
//
//  cudaCheck(hipMalloc((void**)&d_hashGrid, sizeof(HashGrid<float,gpu>)));
//
//  cudaCheck(hipMemcpy(d_hashGrid, &hg, sizeof(HashGrid<float,gpu>), hipMemcpyHostToDevice));
//
//  d_hashGrid->initGrid(hg);
//
//  hashgrid_size<<<1,1>>>(d_hashGrid);
//  hipDeviceSynchronize();
//
//  //d_hashGrid->sortGrid(hg.particleIndices_); 
//  //outputHashGrid<<<1,1>>>(d_hashGrid);
//  hipDeviceSynchronize();
//
//  cudaCheck(hipMalloc((void**)&d_particleWorld, sizeof(ParticleWorld<float,gpu>)));
//  cudaCheck(hipMemcpy(d_particleWorld, &pw, sizeof(ParticleWorld<float,gpu>), hipMemcpyHostToDevice));
//
//  d_particleWorld->initData(pw);
//  hipDeviceSynchronize();
//
//  float jitter = pw.params_->particleRadius_ * 0.01f;
//  unsigned int s = (int) std::ceil(std::pow((float) pw.size_, 1.0f / 3.0f));
//  unsigned int gridSize[3];
//  gridSize[0] = gridSize[1] = gridSize[2] = s;
//  initGrid(gridSize, pw.params_->particleRadius_*2.0f, jitter, pw);
//  hipDeviceSynchronize();
//
//  calcHash(hg, pw);
//  hipDeviceSynchronize();
//
//  d_hashGrid->sortParticles(hg.size_, hg.hashEntries_, hg.particleIndices_);
//  hipDeviceSynchronize();
//
//  output_sorted <<< 1, 1 >>> (d_hashGrid, d_particleWorld);
//  hipDeviceSynchronize();
//
//  reorderDataAndFindCellStart(hg, pw);
//  hipDeviceSynchronize();
//
//  output_cellstart <<< 1, 1 >>>(d_hashGrid);
//  hipDeviceSynchronize();
//
//  collide(hg, pw);
//  hipDeviceSynchronize();
//
//  float timestep = pw.params_->timeStep_;
//
//  integrateSystem(pw.pos_, pw.vel_, timestep, pw.size_);
//  hipDeviceSynchronize();
//
//  test_particleworld<<<1, 1 >>>(d_particleWorld);
//  hipDeviceSynchronize();
//
//}

//void test_hashgrid2(HashGrid<float, cpu> &hg, ParticleWorld<float, cpu> &pw,
//    WorldParameters &params)
//{
//
////  hg.size_ = 10;
////  pw.size_ = hg.size_;
////
////  hg.cellSize_.x = 2.0f * pw.params_->particleRadius_;
////  hg.cellSize_.y = 2.0f * pw.params_->particleRadius_;
////  hg.cellSize_.z = 2.0f * pw.params_->particleRadius_;
////
////  hg.origin_ = pw.params_->origin_;
////
////  hg.gridx_ = pw.params_->gridx_;
////  hg.gridy_ = pw.params_->gridy_;
////  hg.gridz_ = pw.params_->gridz_;
////
////  hg.numCells_ = hg.gridx_ * hg.gridy_ * hg.gridz_;
////
////  cudaCheck(hipMalloc((void**)&d_hashGrid, sizeof(HashGrid<float, gpu>)));
////
////  cudaCheck(hipMemcpy(d_hashGrid, &hg, sizeof(HashGrid<float, gpu>), hipMemcpyHostToDevice));
////
////  d_hashGrid->initGrid(hg);
////
////  cudaCheck(hipMalloc((void**)&d_particleWorld, sizeof(ParticleWorld<float, gpu>)));
////  cudaCheck(hipMemcpy(d_particleWorld, &pw, sizeof(ParticleWorld<float, gpu>), hipMemcpyHostToDevice));
////
////  d_particleWorld->initData(pw);
////
////  float jitter = pw.params_->particleRadius_ * 0.01f;
////  unsigned int s = (int)std::ceil(std::pow((float)pw.size_, 1.0f / 3.0f));
////  unsigned int gridSize[3];
////  gridSize[0] = gridSize[1] = gridSize[2] = s;
////  initGrid(gridSize, pw.params_->particleRadius_*2.0f, jitter, pw);
//
//  calcHash(hg, pw);
//
//  d_hashGrid->sortParticles(hg.size_, hg.hashEntries_, hg.particleIndices_);
//
//  reorderDataAndFindCellStart(hg, pw);
//
////  hipDeviceSynchronize();
////  cudaCheckErrors("reorder test");
////  output_sorted<<<1,1>>>(d_hashGrid, d_particleWorld);
////  output_cellstart<<<1,1>>>(d_hashGrid);
////  hipDeviceSynchronize();
////  return;
//
//  collide(hg, pw);
//
////  output_sorted<<<1,1>>>(d_hashGrid, d_particleWorld);
////  output_cellstart<<<1,1>>>(d_hashGrid);
////  hipDeviceSynchronize();
//  
//  //test_particleworld<<<1, 1 >>>(d_particleWorld);
//  //hipDeviceSynchronize();
//  //cudaCheckErrors("kernel test");
//
//  float timestep = pw.params_->timeStep_;
//
//  integrateSystem(pw.pos_, pw.vel_, timestep, pw.size_);
//  //test_particleworld<<<1, 1 >>>(d_particleWorld);
//  hipDeviceSynchronize();
//
//}

void all_points_dist(UnstructuredGrid<Real, DTraits> &grid)
{

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  d_points_dist << < (grid.nvt_ + 1023) / 1024, 1024 >> > (d_vertices_grid, d_triangles, d_vertices, d_boundingBoxes, d_distance);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  hipDeviceSynchronize();
  printf("GPU time distance: %3.8f [ms]\n", elapsed_time);
  real *mydist = new real[grid.nvt_];
  hipMemcpy(mydist, d_distance, grid.nvt_ * sizeof(real), hipMemcpyDeviceToHost);
  int id = 0;
  for (id = 0; id < grid.nvt_; id++)
  {
    if (grid.m_myTraits[id].iTag)
    {
      grid.m_myTraits[id].distance = -1.0f * sqrtf(mydist[id]);
      grid.m_myTraits[id].distance = sqrtf(mydist[id]);
    }
    else
      grid.m_myTraits[id].distance = sqrtf(mydist[id]);
  }
  hipDeviceSynchronize();

}

__global__ void test_distmap(DistanceMap<float,gpu> *map)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < map->dim_[0]*map->dim_[1])
  {
    vector3 query = map->vertexCoords_[idx];
    query += vector3(0.1,0,0); 
    vector3 cp(0,0,0);
    float dist=0;
    dist=dist+1;
    map->queryMap(query,dist,cp);
  }

}

__global__ void sphere_gpu(UniformGrid<float,ElementCell,VertexTraits<float>,gpu> *g,Sphere<float> *spheres,int s)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < s)
  {
    g->queryVertex(spheres[idx]);
  }

}
__global__ void eval_distmap_kernel(DistanceMap<float, gpu> *map,
    vector3 *v, vector3 *cps, vector3 *normals,
    float *distance, int size,
    TransInfo info)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;


  if (idx < size)
  {
    float dist(1000.0f);
    vector3 query_w = v[idx];
    query_w = info.m2w1 * query_w + info.origin1;
    vector3 query = info.w2m0 * (query_w - info.origin0);
    vector3 cp(0, 0, 0);
    if(!map->boundingBox_.isPointInside(query))
    {
      distance[idx] = 1000.0f;
      return;
      //        printf("vertexCoords = %f %f %f inside\n", v[idx].x, v[idx].y, v[idx].z);               
    }
    map->queryMap(query, dist, cp);
    //      printf("transformed_cpu = %f %f %f = %f\n", query.x, query.y, query.z, dist);               

    //printf("dist : %f v0: %f %f %f\n",dist,info.origin0.x,info.origin0.y,info.origin0.z);
    //printf("cp_on_gpu : %f v0: %f %f %f\n",dist,info.origin0.x,info.origin0.y,info.origin0.z);
    //      for(int j(0); j < 9; ++j)
    //        printf("info.m2w0: %f \n",info.m2w0.m_dEntries[j]);

    // transform the contact point cp into world space
    vector3 c0 = (info.m2w0 * cp) + info.origin0;

    // calculate a normal in world space
    normals[idx] = c0 - query_w;
    normals[idx].normalize();

    // calculate the final contact point as the average
    cp = 0.5f * (c0 + query_w);
    distance[idx] = dist;
    cps[idx] = cp;
    //      else
    //      {
    //        printf("vertexCoords = %f %f %f outside\n", v[idx].x, v[idx].y, v[idx].z);               
    //        map->info();
    //      }
  }
}

void eval_distmap(DistanceMap<float, gpu> *map, vector3 *v, 
    vector3 *cps, vector3 *normals,
    float *distance, int size,
    TransInfo info)
{

  const int tpb = 512;
  int blocks = (size+tpb-1)/tpb;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  eval_distmap_kernel<<< blocks, tpb >>>(map, v, cps, normals, distance, size, info); 
  //eval_distmap_kernel<<< 1, 1 >>>(map, v, cps, normals, distance, size, info); 
  //eval_distmap_kernel<<< 1, 1 >>>(map, v, distance, size, info); 

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  hipDeviceSynchronize();
  printf("> Elapsed time gpu distmap: %3.8f [ms].\n", elapsed_time);
  cudaCheckErrors("eval_distmap");
}

__global__ void dmap_kernel(UniformGrid<float,ElementCell,VertexTraits<float>,gpu> *g,
    DistanceMap<float,gpu> *map, vector3 com, Mat3f m)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < map->dim_[0]*map->dim_[1])
  {
    vector3 query = g->traits_.vertexCoords_[idx] - com;
    query = m * query;
    vector3 cp(0,0,0);
    g->traits_.fbmVertices_[idx] = map->queryFBM(query);
  }

}

__global__ void queryGrid(UniformGrid<float,ElementCell,VertexTraits<float>,gpu> *g, int j)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(g->traits_.fbmVertices_[j])
    printf("fbm_vertex = %i %i \n", j, g->traits_.fbmVertices_[j]);

}

__global__ void copyFBM(UniformGrid<float,ElementCell,VertexTraits<float>,gpu> *g, int *d, int size)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < size)
  {
    d[idx] = g->traits_.fbmVertices_[idx];
  }

}

void sphere_test(RigidBody *body, UniformGrid<Real,ElementCell,VertexTraits<Real>> &grid)
{

  int size = body->map_->dim_[0] * body->map_->dim_[1];

  vector3 *testVectors = new vector3[NN];
  vector3 *d_testVectors;

  Real *distance_res = new Real[NN];
  float *d_distance_res;
  float *distance_gpu = new float[NN];

  std::vector<Sphere<float>> spheres;

  for(int i=0; i < NN; i++)
  {
    vector3 vr(0,0,0);

    vr.x = -body->map_->boundingBox_.extents_[0] + frand() *
      (2.0 * body->map_->boundingBox_.extents_[0]); 

    vr.y = -body->map_->boundingBox_.extents_[1] + frand() *
      (2.0 * body->map_->boundingBox_.extents_[1]); 

    vr.z = -body->map_->boundingBox_.extents_[2] + frand() *
      (2.0 * body->map_->boundingBox_.extents_[2]); 

    testVectors[i] = vr;
  }

  hipMalloc((void**)&(d_testVectors), NN*sizeof(vector3));
  hipMemcpy(d_testVectors, testVectors, NN*sizeof(vector3), hipMemcpyHostToDevice);

  hipMalloc((void**)&(d_distance_res), NN*sizeof(float));

  int vx = grid.traits_.cells_[0]+1;
  int vxy=vx*vx*vx;
  printf("CPU distmap for %i points\n", vxy);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  test_dist<<< (vxy+255)/256, 256 >>>(body->map_gpu_,d_testVectors, d_distance_res);
  hipMemcpy(distance_gpu, d_distance_res, NN*sizeof(float), hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  hipDeviceSynchronize();
  printf("Elapsed time gpu distmap: %3.8f [ms].\n", elapsed_time);
  float gpu_distmap = elapsed_time;

  CPerfTimer timer;
  timer.Start();
  //  for (int i = 0; i < vxy; i++)
  //  {
  //    Vector3<Real> v=grid.traits_.vertexCoords_[i];
  //    std::pair<Real, Vector3<Real>> res = body->map_->queryMap(v);
  //    grid.traits_.fbmVertices_[i] = (res.first < 0.0) ? 1 : 0;
  //  }

  float cpu_distmap = timer.GetTime();
  std::cout << "Elapsed time cpu distmap: " <<  cpu_distmap << " [ms]." << std::endl;

  hipDeviceSynchronize();

  grid.queryVertex(body->spheres[185]);

  printf("Inner sphere test with %i spheres\n", int(body->spheres.size()));

  timer.Start();

  //  for(auto &sphere : body->spheres)
  //  {
  //    grid.query(sphere);
  //  }

  float cpu_spheres = timer.GetTime();
  std::cout << "Elapsed time cpu spheres: " <<  cpu_spheres << " [ms]." << std::endl;

  for (int i = 0; i < vxy; i++)
  {
    grid.traits_.fbmVertices_[i] = 0;
  }

  timer.Start();

  for(auto &sphere : body->spheres)
  {
    grid.queryVertex(sphere);
  }

  float cpu_spheres_vertex = timer.GetTime();
  std::cout << "Elapsed time cpu spheres[vertex]: " <<  cpu_spheres_vertex << " [ms]." << std::endl;

  for(auto &sphere : body->spheres)
  {
    Sphere<float> s(sphere); 
    spheres.push_back(s);
  }

  Sphere<float> *dev_spheres;
  hipMalloc((void**)&(dev_spheres), spheres.size() * sizeof(Sphere<float>));
  hipMemcpy(dev_spheres, spheres.data(), spheres.size()*sizeof(Sphere<float>), hipMemcpyHostToDevice);

  hipEvent_t start0, stop0;
  hipEventCreate(&start0);
  hipEventCreate(&stop0);
  hipEventRecord(start0, 0);

  sphere_gpu<<< (spheres.size()+255)/256, 256 >>>(d_unigrid_gpu,dev_spheres,spheres.size());

  hipEventRecord(stop0, 0);
  hipEventSynchronize(stop0);
  float elapsed_time0;
  hipEventElapsedTime(&elapsed_time0, start0, stop0);
  hipDeviceSynchronize();
  printf("Elapsed time gpu spheres[vertex]: %3.8f [ms].\n", elapsed_time0);

  int size2 = (grid.m_iDimension[0]+1) * (grid.m_iDimension[1]+1) * (grid.m_iDimension[2]+1);
  int *fbmVertices = new int[size2];
  int *dev_fbmVertices;
  hipMalloc((void**)&(dev_fbmVertices), size2 * sizeof(int));

  copyFBM<<< (size2+255)/256, 256 >>>(d_unigrid_gpu,dev_fbmVertices,size2);

  hipMemcpy(fbmVertices, dev_fbmVertices,
      size2 * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  int gpuIn=0;
  int cpuIn=0;
  for (int i = 0; i < size2; i++)
  {
    if(fbmVertices[i] != grid.traits_.fbmVertices_[i])
    {
      printf("gpu = %i cpu = %i %i\n", fbmVertices[i], grid.traits_.fbmVertices_[i],i);
    }
    if(fbmVertices[i])
    {
      gpuIn++;
    }
    if(grid.traits_.fbmVertices_[i])
    {
      cpuIn++;
    }
  }

  printf("gpuIn = %i \n", gpuIn);
  printf("cpuIn = %i \n", cpuIn);

  printf("cpu_distmap = %3.8f \n", cpu_distmap);
  printf("gpu_distmap = %3.8f \n", gpu_distmap);
  printf("cpu_spheres = %3.8f \n", cpu_spheres_vertex);
  printf("gpu_distmap = %3.8f \n", elapsed_time0);

  delete[] testVectors;
  delete[] distance_res;
  delete[] distance_gpu;
  delete[] fbmVertices;

  hipFree(d_testVectors);
  hipFree(d_distance_res);
  hipFree(dev_fbmVertices);

}

__global__ void test_grid(UniformGrid<float,ElementCell,VertexTraits<float>,gpu> *g,
    int dimx, int dimy, int dimz)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  {
    if(dimx == g->dim_[0])
    {
      printf("> Test 1: size OK!\n");
    }
    else
    {
      printf("> Test 1: size not OK, failed!\n");
    }

    if(dimx*dimy*dimz == g->dim_[0]*g->dim_[1]*g->dim_[2])
    {
      printf("> Test 2: total size OK!\n");
    }
    else
    {
      printf("> Test 1: total size not OK, failed!\n");
    }

  }

}

void transfer_uniformgrid(UniformGrid<Real,ElementCell,VertexTraits<Real>> *grid)
{

  UniformGrid<float,ElementCell,VertexTraits<float>,cpu> grid_(grid);

  hipMalloc((void**)&(d_unigrid_gpu), sizeof(UniformGrid<float,ElementCell,VertexTraits<float>,gpu>));
  cudaCheckErrors("Allocate uniform grid");

  hipMemcpy(d_unigrid_gpu, &grid_, sizeof(UniformGrid<float,ElementCell,VertexTraits<float>,cpu>), hipMemcpyHostToDevice);
  cudaCheckErrors("copy uniformgrid class");

  d_unigrid_gpu->transferData(grid_);

  test_grid<<<1,1>>>(d_unigrid_gpu, grid->m_iDimension[0],
      grid->m_iDimension[1],
      grid->m_iDimension[2]);
  hipDeviceSynchronize();

}

__global__ void dmap_kernel_test(DistanceMap<float,gpu> *map, UniformGrid<float,ElementCell,VertexTraits<float>,gpu> *g)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < map->dim_[0]*map->dim_[1])
  {
    vector3 query(0,0.05,0);
    vector3 cp(0,0,0);
    float dist=0;
    //printf("fbm_vertex = %i %i \n", j, g->traits_.fbmVertices_[j]);
    vector3 v = g->traits_.vertexCoords_[16000];
    printf(" vertexCoords[16000] = %f %f %f\n", v.x, v.y, v.z);
    int result = map->queryFBM(query);
    printf("idx = %d vertexCoords = %f %f %f = %d\n", idx, query.x, query.y, query.z, result);               
  }

}

void allocate_dmap(RigidBody* body)
{

  DistanceMap<float, cpu> map_(body->map_);

  hipMalloc((void**)&(body->map_gpu_), sizeof(DistanceMap<float, gpu>));
  cudaCheckErrors("Allocate dmap");

  hipMemcpy(body->map_gpu_, &map_, sizeof(DistanceMap<float, cpu>), hipMemcpyHostToDevice);
  cudaCheckErrors("copy distancemap class");

  body->map_gpu_->transferData(map_);

  gpu_map_test <<<1, 1>>>(body->map_gpu_);

  //dist_comp<<<1,1>>>(body->map_gpu_);

  hipDeviceSynchronize();

  std::pair<Real, Vector3<Real> > result0 = body->map_->queryMap(VECTOR3(0.001,0,0));

  printf("map0: %f\n", result0.first);

}

void allocate_distancemaps(std::vector<RigidBody*> &rigidBodies, std::vector<DistanceMap<Real>* > &maps, std::vector<int> &bodyToMap)
{

  std::vector< DistanceMap<float,gpu>* > gpu_maps;

  for(auto &map : maps)
  {
    DistanceMap<float, cpu> map_(map);

    DistanceMap<float,gpu> *map_gpu;

    hipMalloc((void**)&(map_gpu), sizeof(DistanceMap<float, gpu>));
    cudaCheckErrors("Allocate dmap");

    hipMemcpy(map_gpu, &map_, sizeof(DistanceMap<float, cpu>), hipMemcpyHostToDevice);
    cudaCheckErrors("copy distancemap class");

    map_gpu->transferData(map_);

    std::pair<Real, Vector3<Real> > result0 = map->queryMap(VECTOR3(0.001,0,0));

    test_dist_comp<<<1,1>>>(map_gpu, result0.first);
    hipDeviceSynchronize();

    gpu_maps.push_back(map_gpu);
  }

  for (unsigned i=0; i < rigidBodies.size(); ++i)
  {
    RigidBody *body = rigidBodies[i];

    body->map_gpu_ = gpu_maps[bodyToMap[i]];
  }

  hipDeviceSynchronize();

}

void transfer_distancemap(RigidBody *body, DistanceMap<float,cpu> *map)
{

  DistanceMap<float,cpu> &map_=*map;

  hipMalloc((void**)&(body->map_gpu_), sizeof(DistanceMap<float,gpu>));
  cudaCheckErrors("Allocate dmap");

  hipMemcpy(body->map_gpu_, &map_, sizeof(DistanceMap<float,cpu>), hipMemcpyHostToDevice);
  cudaCheckErrors("copy distancemap class");

  body->map_gpu_->transferData(map_);
  hipDeviceSynchronize();

}

void copy_distancemap(DistanceMap<Real,cpu> *map)
{

  DMap map_;

  map_.dim_[0] = map->dim_[0];
  map_.dim_[1] = map->dim_[1];

  map_.cells_[0] = map->cells_[0];
  map_.cells_[1] = map->cells_[1];
  map_.cells_[2] = map->cells_[2];

  map_.cellSize_ = map->cellSize_; 

  Vector3<float> vmin, vmax;
  vmin.x = (float)map->boundingBox_.vertices_[0].x;
  vmin.y = (float)map->boundingBox_.vertices_[0].y;
  vmin.z = (float)map->boundingBox_.vertices_[0].z;

  vmax.x = (float)map->boundingBox_.vertices_[1].x;
  vmax.y = (float)map->boundingBox_.vertices_[1].y;
  vmax.z = (float)map->boundingBox_.vertices_[1].z;

  map_.bv_.init(vmin, vmax);

  hipMalloc((void**)&d_map, sizeof(DMap));
  cudaCheckErrors("Allocate dmap");

  hipMemcpy(d_map, &map_, sizeof(DMap), hipMemcpyHostToDevice);
  cudaCheckErrors("copy distancemap class");

  Vector3<float> *vertexCoords;
  Vector3<float> *normals;
  Vector3<float> *contactPoints;      

  float *distance_;

  int size = map->dim_[0] * map->dim_[1]; 

  map->outputInfo();

  vertexCoords = new Vector3<float>[size];
  normals = new Vector3<float>[size];
  contactPoints = new Vector3<float>[size];
  distance_ = new float[size];

  for (int i = 0; i < size; i++)
  {
    vertexCoords[i].x = (float)map->vertexCoords_[i].x;
    vertexCoords[i].y = (float)map->vertexCoords_[i].y;
    vertexCoords[i].z = (float)map->vertexCoords_[i].z;

    normals[i].x = (float)map->normals_[i].x;
    normals[i].y = (float)map->normals_[i].y;
    normals[i].z = (float)map->normals_[i].z;

    contactPoints[i].x = (float)map->contactPoints_[i].x;
    contactPoints[i].y = (float)map->contactPoints_[i].y;
    contactPoints[i].z = (float)map->contactPoints_[i].z;

    distance_[i] = (float)map->distance_[i];
  }

  hipMalloc((void**)&d_vertexCoords, size * sizeof(vector3));
  cudaCheckErrors("Allocate vertices distancemap");

  hipMemcpy(d_vertexCoords, vertexCoords, size * sizeof(vector3), hipMemcpyHostToDevice);
  cudaCheckErrors("copy vertices distance");

  hipMemcpy(&d_map->vertices_ , &d_vertexCoords, sizeof(vector3*), hipMemcpyHostToDevice);
  cudaCheckErrors("copy vertices distance");

  hipMalloc((void**)&d_normals, size * sizeof(vector3));
  cudaCheckErrors("Allocate vertices normals");

  hipMemcpy(d_normals, normals, size * sizeof(vector3), hipMemcpyHostToDevice);
  cudaCheckErrors("copy vertices normals");

  hipMemcpy(&d_map->normals_ , &d_normals, sizeof(vector3*), hipMemcpyHostToDevice);
  cudaCheckErrors("copy vertices normals");

  hipMalloc((void**)&d_contactPoints, size * sizeof(vector3));
  cudaCheckErrors("Allocate vertices contactPoints");

  hipMemcpy(d_contactPoints, contactPoints, size * sizeof(vector3), hipMemcpyHostToDevice);
  cudaCheckErrors("copy vertices contactPoints");

  hipMemcpy(&d_map->contactPoints_ , &d_contactPoints, sizeof(vector3*), hipMemcpyHostToDevice);
  cudaCheckErrors("copy vertices contactPoints");

  hipMalloc((void**)&d_distance_map, size * sizeof(float));
  cudaCheckErrors("Allocate distance");

  hipMemcpy(d_distance_map, distance_, size * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("copy distance");

  hipMemcpy(&d_map->distance_ , &d_distance_map, sizeof(float*), hipMemcpyHostToDevice);
  cudaCheckErrors("copy distance");

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  test_distmap<<<(size+255)/256, 256 >>>(d_map, d_vertexCoords);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  hipDeviceSynchronize();
  printf("GPU distmap coll: %3.8f [ms]\n", elapsed_time);

  hipDeviceSynchronize();

  //  std::pair<Real,Vector3<Real>> res = map->queryMap(map->vertexCoords_[0]+Vector3<Real>(0.1,0,0));
  //  std::cout << "query_cpu" << map->vertexCoords_[0] << std::endl;
  //  std::cout << "cp" << res.second << std::endl;
  //  std::cout << "dist_cpu" << res.first << std::endl;
  //  std::cout << "dist[100]" <<  map->distance_[100] << std::endl;
  //  exit(0);

  CPerfTimer timer;
  timer.Start();
  for (int i = 0; i < size; i++)
  {
    map->queryMap(map->vertexCoords_[i]);
  }

  std::cout << "Elapsed time gpu[ms]:" <<  timer.GetTime() * 1000.0 << std::endl;
  delete[] vertexCoords;
  delete[] normals;
  delete[] contactPoints;
  delete[] distance_;

}

void copy_mesh(Model3D *model){

  int nVertices = 0;

  g_model = model;

  vector3  *meshVertices;

  for (auto &mesh : model->meshes_)
  {
    nVertices += mesh.numVerts_;
    meshVertices = (vector3*)malloc(sizeof(vector3)*mesh.numVerts_);
    for (int i = 0; i < mesh.vertices_.Size(); i++)
    {
      meshVertices[i].x = (real)mesh.vertices_[i].x;
      meshVertices[i].y = (real)mesh.vertices_[i].y;
      meshVertices[i].z = (real)mesh.vertices_[i].z;
    }
  }

  printf("Number of triangles: %i\n",nVertices);
  g_triangles = nVertices;

  hipMalloc((void**)&d_vertices, nVertices * sizeof(vector3));
  cudaCheckErrors("Allocate vertices");

  hipMemcpy(d_vertices, meshVertices, nVertices * sizeof(vector3), hipMemcpyHostToDevice);
  cudaCheckErrors("Copy vertices");
  hipDeviceSynchronize();

  hipMemcpyToSymbol(HIP_SYMBOL(d_nVertices), &nVertices, sizeof(int));
  cudaCheckErrors("Copy number of vertices");

  free(meshVertices);

  hipDeviceSynchronize();

}

void my_cuda_func(Model3D *model, UnstructuredGrid<Real, DTraits> &grid){

  int nTriangles = 0;
  int nVertices = 0;

  g_model = model;

  triangle *meshTriangles;
  vector3  *meshVertices;

  for (auto &mesh : model->meshes_)
  {
    nTriangles += mesh.numFaces_;
    meshTriangles=(triangle*)malloc(sizeof(triangle)*mesh.numFaces_);
    for (int i = 0; i < mesh.faces_.Size(); i++)
    {
      meshTriangles[i].idx0 = mesh.faces_[i][0];
      meshTriangles[i].idx1 = mesh.faces_[i][1];
      meshTriangles[i].idx2 = mesh.faces_[i][2];
    }

    nVertices += mesh.numVerts_;
    meshVertices = (vector3*)malloc(sizeof(vector3)*mesh.numVerts_);
    for (int i = 0; i < mesh.vertices_.Size(); i++)
    {
      meshVertices[i].x = (real)mesh.vertices_[i].x;
      meshVertices[i].y = (real)mesh.vertices_[i].y;
      meshVertices[i].z = (real)mesh.vertices_[i].z;
    }
  }

  model->meshes_[0].generateTriangleBoundingBoxes();

  AABB3f *boxes = new AABB3f[nTriangles];
  hipMalloc((void**)&d_boundingBoxes, sizeof(AABB3f)* nTriangles);
  for (int i = 0; i < nTriangles; i++)
  {
    vector3 vmin, vmax;
    vmin.x = (real)model->meshes_[0].triangleAABBs_[i].vertices_[0].x;
    vmin.y = (real)model->meshes_[0].triangleAABBs_[i].vertices_[0].y;
    vmin.z = (real)model->meshes_[0].triangleAABBs_[i].vertices_[0].z;

    vmax.x = (real)model->meshes_[0].triangleAABBs_[i].vertices_[1].x;
    vmax.y = (real)model->meshes_[0].triangleAABBs_[i].vertices_[1].y;
    vmax.z = (real)model->meshes_[0].triangleAABBs_[i].vertices_[1].z;

    boxes[i].init(vmin, vmax);
  }

  hipMemcpy(d_boundingBoxes, boxes, nTriangles * sizeof(AABB3f), hipMemcpyHostToDevice);

  delete[] boxes;

  printf("Number of triangles: %i\n",nTriangles);
  g_triangles = nTriangles;
  hipMalloc((void**)&d_triangles, nTriangles * sizeof(triangle));
  cudaCheckErrors("Allocate triangles");

  hipMemcpy(d_triangles, meshTriangles, nTriangles * sizeof(triangle),hipMemcpyHostToDevice);
  cudaCheckErrors("Copy triangles");

  hipMemcpyToSymbol(HIP_SYMBOL(d_nTriangles), &nTriangles, sizeof(int));
  cudaCheckErrors("Copy number of triangles");

  printf("CPU: Triangle[52].idx0 = %i \n", meshTriangles[52].idx0);

  hipMalloc((void**)&d_vertices, nVertices * sizeof(vector3));
  cudaCheckErrors("Allocate vertices");

  hipMalloc((void**)&d_inout, grid.nvt_ * sizeof(int));
  cudaCheckErrors("Allocate vertex traits");

  hipMemcpy(d_vertices, meshVertices, nVertices * sizeof(vector3), hipMemcpyHostToDevice);
  cudaCheckErrors("Copy vertices");
  hipDeviceSynchronize();

  hipMemcpyToSymbol(HIP_SYMBOL(d_nVertices), &nVertices, sizeof(int));
  cudaCheckErrors("Copy number of vertices");

  printf("CPU: Number of vertices: %i\n", nVertices);
  printf("CPU: Vertex[52].y = %f \n", meshVertices[52].y);

  free(meshTriangles);

  free(meshVertices);

  hipMalloc((void**)&d_vertices_grid, grid.nvt_ * sizeof(vector3));
  cudaCheckErrors("Allocate grid vertices");

  meshVertices = (vector3*)malloc(sizeof(vector3)*grid.nvt_);
  for (int i = 0; i < grid.nvt_; i++)
  {
    meshVertices[i].x = (real)grid.vertexCoords_[i].x;
    meshVertices[i].y = (real)grid.vertexCoords_[i].y;
    meshVertices[i].z = (real)grid.vertexCoords_[i].z;
  }

  hipMemcpy(d_vertices_grid, meshVertices, grid.nvt_ * sizeof(vector3), hipMemcpyHostToDevice);
  cudaCheckErrors("Copy grid vertices");
  hipDeviceSynchronize();

  hipMemcpyToSymbol(HIP_SYMBOL(d_nVertices_grid), &grid.nvt_, sizeof(int));
  g_verticesGrid = grid.nvt_;
  free(meshVertices);

  hipMalloc((void**)&d_distance, grid.nvt_ * sizeof(real));
  cudaCheckErrors("Allocation for distance array");

  hipMemset(d_distance, 0, grid.nvt_ * sizeof(real));

}

void allocateNodes(std::list<int> *triangleIdx, AABB3f *boxes, int *pSize, int nNodes)
{

  hipMalloc((void**)&d_nodes, nNodes * sizeof(BVHNode<float>));
  cudaCheckErrors("Allocate nodes");

  for (int i = 0; i < nNodes; i++)
  {
    hipMemcpy(&d_nodes[i].nTriangles_, &pSize[i], sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&(d_nodes[i].bv_), &boxes[i], sizeof(AABB3f), hipMemcpyHostToDevice);
  }

  int **d_indices = new int*[nNodes];

  for (int i = 0; i < nNodes; i++)
  {
    hipMalloc((void**)&d_indices[i], pSize[i] * sizeof(int));
    hipMemcpy(&d_nodes[i].indices_, &d_indices[i], sizeof(int*), hipMemcpyHostToDevice);
  }

  hipDeviceSynchronize();

  int tsize = triangleIdx[0].size();

  printf("nodes = %i, psize[0] = %i %i \n", nNodes, pSize[0],tsize);
  int indices[10000];
  int j = 0;

  for (auto &idx : triangleIdx[0])
  {
    indices[j] = idx;
    j++;
  }
  printf("CPU: nodes[1].indices_[0] = %i \n", indices[0]);

  for (int i = 0; i < nNodes; i++)
  {

    int j = 0;
    for (auto &idx : triangleIdx[i])
    {
      indices[j] = idx;
      j++;
    }
    if (i==0)
      printf("CPU: nodes[1].indices_[0] = %i \n", indices[0]);

    hipMemcpy(d_indices[i], indices, sizeof(int) * pSize[i], hipMemcpyHostToDevice);
    hipDeviceSynchronize();
  }

  printf("gpu triangles = %i \n", pSize[1]);
  printf("center = %f \n", boxes[1].center_.x);

}

void query_uniformgrid(RigidBody *body, UniformGrid<Real,ElementCell,VertexTraits<Real>> &grid)
{

  int size(body->map_->dim_[0] * body->map_->dim_[1]);

  int vx(grid.traits_.cells_[0]+1);
  int vxy(vx*vx*vx);

  hipDeviceSynchronize();
  vector3 com(body->com_.x, body->com_.y, body->com_.z);

  Mat3 m(body->matTransform_);
  m.TransposeMatrix();

  Mat3f myMat;
  myMat.m_d00 = m.m_d00;
  myMat.m_d01 = m.m_d01;
  myMat.m_d02 = m.m_d02;

  myMat.m_d10 = m.m_d10;
  myMat.m_d11 = m.m_d11;
  myMat.m_d12 = m.m_d12;

  myMat.m_d20 = m.m_d20;
  myMat.m_d21 = m.m_d21;
  myMat.m_d22 = m.m_d22;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dmap_kernel<<< (vxy+255)/256, 256 >>>(d_unigrid_gpu, body->map_gpu_, com, myMat);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipDeviceSynchronize();

  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  hipDeviceSynchronize();

  float gpu_distmap = elapsed_time;

  int size2 = (grid.m_iDimension[0]+1) * (grid.m_iDimension[1]+1) * (grid.m_iDimension[2]+1);

  std::vector<int> fbmVertices(size2);
  int *dev_fbmVertices;

  hipMalloc((void**)&(dev_fbmVertices), size2 * sizeof(int));

  copyFBM<<< (size2+255)/256, 256 >>>(d_unigrid_gpu,dev_fbmVertices,size2);

  hipMemcpy(fbmVertices.data(),
      dev_fbmVertices,
      size2 * sizeof(int),
      hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  int inside(0);

  for(int i(0); i < size2; ++i)
  {
    if (fbmVertices[i])
    {
      inside++;
      grid.traits_.fbmVertices_[i] = fbmVertices[i];
    }
  }

  int inside_cpu(0);
  for (int i(0); i < size2; ++i)
  {     
    std::pair<Real, Vector3<Real>> res = body->map_->queryMap(grid.traits_.vertexCoords_[i]);
    if (res.first < 0.0) inside_cpu++;
  }

  test_inside(inside_cpu, inside);
  printf("> Elapsed time gpu distmap: %3.8f [ms].\n", elapsed_time);

  hipFree(dev_fbmVertices);

}

void cleanGPU()
{

  hipFree(d_triangles);
  hipFree(d_vertices);
  hipFree(d_vertices_grid);
  hipFree(d_inout);
  hipFree(d_distance);
  hipFree(d_nodes);

}

